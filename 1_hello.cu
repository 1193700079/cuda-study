
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void helloFromGPU(){

printf("Hello World from GPU\n");

printf("blockDim:x=%d,y=%d,z=%d,gridDim:x=%d,y=%d,z=%d Current threadIdx=%d,y=%d,z=%d\n",blockDim.x,blockDim.y,blockDim.z,gridDim.x,gridDim.y,gridDim.z,threadIdx.x,threadIdx.y,threadIdx.z);
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU\n");

    dim3 grid;
    grid.x = 2;
    grid.y = 2;
    dim3 block;
    block.x = 2;
    block.y = 2;
    helloFromGPU<<<grid,block>>>();
    hipDeviceReset();
    return 0;
}
//sudo apt-get -o Acquire::http::proxy="http://127.0.0.1:7890/" install gcc-7 g++-7 -y