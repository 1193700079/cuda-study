
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
__global__ void generate_random_numbers(float *numbers, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n)
    {
        hiprandState state;
        hiprand_init(clock64(), idx, 0, &state);
        numbers[idx] = hiprand_uniform(&state);
    }
}

int main()
{
    int n = 10;
    float *numbers;
    hipMalloc(&numbers, n * sizeof(float));

    generate_random_numbers<<<1, n>>>(numbers, n);

    float *host_numbers = new float[n];
    hipMemcpy(host_numbers, numbers, n * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i)
    {
        // printf("%f\n", host_numbers[i]);
        std::cout<<host_numbers[i]<<std::endl;
    }

    delete[] host_numbers;
    hipFree(numbers);

    return 0;
}